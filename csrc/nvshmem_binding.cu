#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Perplexity AI
 */
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Exception.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <torch/library.h>

#include <cstdint>
#include <cstdlib>
#include <string>
#include <vector>

#define NVSHMEMCHECK(stmt)                                                                    \
  do {                                                                                        \
    int result = (stmt);                                                                      \
    if (NVSHMEMX_SUCCESS != result) {                                                         \
      fprintf(stderr, "[%s:%d] nvshmem failed with error %d \n", __FILE__, __LINE__, result); \
      exit(-1);                                                                               \
    }                                                                                         \
  } while (0)

namespace {

static std::unordered_map<std::string, nvshmem_team_t> group_name_to_team_;

nvshmem_team_t group_to_team(
    const std::string& group_name,
    const std::vector<int>& global_ranks) {
  auto it = group_name_to_team_.find(group_name);
  if (it != group_name_to_team_.end()) {
    return it->second;
  }
}

int create_team(const std::string& group_name, const std::vector<int>& global_ranks) {
  nvshmem_team_config_t *config;
  config = (nvshmem_team_config_t *)malloc(sizeof(nvshmem_team_config_t));
  *config = NVSHMEM_TEAM_CONFIG_INITIALIZER;

  int status = 0;
  int my_pe = nvshmem_my_pe();
  int npes = nvshmem_n_pes();
  int team_size = 0;
  int my_idx_in_team;

  nvshmem_team_t team;
  NVSHMEM_CHECK(
      nvshmemx_team_init(
          team,
          config, 
          NVSHMEM_TEAM_CONFIG_MASK_UNIQUEID,
          team_size,
          my_idx_in_team),
          "nvshmemx_team_init failed");
  group_name_to_team_[group_name] = team;
  TORCH_CHECK(team != NVSHMEM_TEAM_INVALID);
}

at::Tensor get_team_unique_id() {
  nvshmemx_team_uniqueid_t uid; // should this be *
  nvshmemx_team_get_uniqueid(&uid); // should this be non and
  return at::from_blob(&uid, sizeof(uid), at::kByte).clone();
}

int64_t team_unique_id_size() { return sizeof(nvshmemx_team_uniqueid_t); }


at::Tensor get_unique_id() {
  nvshmemx_uniqueid_t uid = NVSHMEMX_UNIQUEID_INITIALIZER;
  nvshmemx_get_uniqueid(&uid);
  return at::from_blob(&uid, sizeof(uid), at::kByte).clone();
}

int64_t unique_id_size() { return sizeof(nvshmemx_uniqueid_t); }

int64_t init(at::Tensor uid, int64_t rank, int64_t world_size) {
  TORCH_CHECK(uid.device().is_cpu(), "uid must be a CPU tensor");
  TORCH_CHECK(uid.scalar_type() == at::kByte, "uid must be a byte tensor");
  TORCH_CHECK(uid.numel() == sizeof(nvshmemx_uniqueid_t),
              "Invalid unique id size. Expected: ", sizeof(nvshmemx_uniqueid_t),
              ", Got: ", uid.numel(), ")");
  nvshmemx_uniqueid_t id;
  std::memcpy(&id, uid.data_ptr(), sizeof(id));
  nvshmemx_init_attr_t attr = NVSHMEMX_INIT_ATTR_INITIALIZER;
  nvshmemx_set_attr_uniqueid_args(rank, world_size, &id, &attr);
  return nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr);
}

void finalize() { nvshmem_finalize(); }

int64_t my_pe() { return nvshmem_my_pe(); }

int64_t n_pes() { return nvshmem_n_pes(); }

at::Tensor malloc_tensor(const std::vector<int64_t>& shape, c10::ScalarType dtype,
                         const c10::Device& device) {
  size_t size = c10::elementSize(dtype) * c10::multiply_integers(shape);
  void* ptr = nvshmem_malloc(size);
  if (ptr == nullptr) {
    AT_ERROR("nvshmem_malloc failed. size: ", size);
  }
  return at::from_blob(
      ptr, shape, [](void* ptr) { nvshmem_free(ptr); },
      at::TensorOptions().dtype(dtype).device(device));
}

int64_t multicast_ptr(at::Tensor tensor, const std::string& group_name) {
  auto team = group_to_team(group_name);
  void *mc_ptr = nvshmemx_mc_ptr(team, (void *) tensor.data_ptr());
  if (mc_ptr == nullptr) {
    AT_ERROR("nvshmemx_mc_ptr failed.");
  }
  return reinterpret_cast<int64_t>(mc_ptr);
}

void barrier_all() { nvshmem_barrier_all(); }

void barrier_all_on_current_stream() {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_barrier_all_on_stream(stream);
}

void alltoall(at::Tensor dest, at::Tensor source, const std::string& group_name) {
  TORCH_CHECK(dest.is_contiguous(), "dest must be contiguous");
  TORCH_CHECK(source.is_contiguous(), "source must be contiguous");

  size_t nbytes = dest.numel() * dest.itemsize() / dest.size(0);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto team = group_to_team(group_name);
  NVSHMEMCHECK(nvshmemx_alltoallmem_on_stream(team, (uint8_t*)dest.data_ptr(),
                                              (uint8_t*)source.data_ptr(), nbytes, stream));
}

void fake_alltoall(at::Tensor dest, at::Tensor source, const std::string& group_name) {}

void sum_reduce(at::Tensor dest, at::Tensor source, int64_t nelems, const std::string& group_name) {
  TORCH_CHECK(dest.is_contiguous(), "dest must be contiguous");
  TORCH_CHECK(source.is_contiguous(), "source must be contiguous");
  TORCH_CHECK(dest.scalar_type() == source.scalar_type(),
              "dest and source must have the same dtype");

  // Add validation and conversion
  TORCH_CHECK(nelems >= 0, "nelems must be non-negative, got ", nelems);
  TORCH_CHECK(nelems <= SIZE_MAX, "nelems too large: ", nelems, " > ", SIZE_MAX);
  size_t nelems_size_t = static_cast<size_t>(nelems);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto team = group_to_team(group_name);
  switch (dest.scalar_type()) {
    case at::kHalf:  // float16
      NVSHMEMCHECK(nvshmemx_half_sum_reduce_on_stream(team, (__half*)dest.data_ptr(),
                                                      (__half*)source.data_ptr(), nelems_size_t,
                                                      stream));
      break;
    case at::kFloat:  // float32
      NVSHMEMCHECK(nvshmemx_float_sum_reduce_on_stream(team, (float*)dest.data_ptr(),
                                                       (float*)source.data_ptr(), nelems_size_t,
                                                       stream));
      break;
    case at::kBFloat16:  // bfloat16
      NVSHMEMCHECK(nvshmemx_bfloat16_sum_reduce_on_stream(
          team, (__hip_bfloat16*)dest.data_ptr(), (__hip_bfloat16*)source.data_ptr(),
          nelems_size_t, stream));
      break;

    default:
      TORCH_CHECK(false, "Unsupported dtype for nvshmem_sum_reduce: ", dest.scalar_type());
  }
}

void fake_sum_reduce(at::Tensor dest, at::Tensor source, int64_t nelems, const std::string& group_name) {}

void allreduce_on_stream_with_copy(at::Tensor dest_symm, at::Tensor source_symm,
                                   at::Tensor dest_local, at::Tensor source_local, int64_t nelems, const std::string& group_name) {
  TORCH_CHECK(dest_symm.is_contiguous(), "dest_symm must be contiguous");
  TORCH_CHECK(source_symm.is_contiguous(), "source_symm must be contiguous");
  TORCH_CHECK(dest_local.is_contiguous(), "dest_local must be contiguous");
  TORCH_CHECK(source_local.is_contiguous(), "source_local must be contiguous");
  TORCH_CHECK(dest_symm.scalar_type() == source_symm.scalar_type(),
              "dest_symm and source_symm must have the same dtype");
  TORCH_CHECK(dest_symm.scalar_type() == source_local.scalar_type(),
              "dest_symm and source_local must have the same dtype");
  TORCH_CHECK(dest_local.scalar_type() == source_local.scalar_type(),
              "dest_local and source_local must have the same dtype");

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto team = group_to_team(group_name);
  hipMemcpyAsync(source_symm.data_ptr(), source_local.data_ptr(),
                  nelems * source_local.element_size(), hipMemcpyDefault, stream);
  nvshmemx_barrier_on_stream(team, stream);
  sum_reduce(dest_symm, source_symm, nelems, group_name);
  hipMemcpyAsync(dest_local.data_ptr(), dest_symm.data_ptr(), nelems * dest_local.element_size(),
                  hipMemcpyDefault, stream);
  hipStreamSynchronize(stream);
}

void fake_allreduce_on_stream_with_copy(at::Tensor dest_symm, at::Tensor source_symm,
                                        at::Tensor dest_local, at::Tensor source_local,
                                        int64_t nelems) {}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("nvshmem_get_unique_id", &get_unique_id);
  m.def("nvshmem_unique_id_size", &unique_id_size);
  m.def("nvshmem_team_unique_id_size", &team_unique_id_size);
  m.def("nvshmem_get_team_unique_id", &get_team_unique_id);
  m.def("nvshmem_init", &init);
  m.def("nvshmem_finalize", &finalize);
  m.def("nvshmem_my_pe", &my_pe);
  m.def("nvshmem_n_pes", &n_pes);
  m.def("nvshmem_malloc", &malloc_tensor);
  m.def("nvshmem_multicast_ptr", &multicast_ptr);
  m.def("nvshmem_barrier_all", &barrier_all);
  m.def("nvshmem_barrier_all_on_current_stream", &barrier_all_on_current_stream);
  m.def("nvshmem_alltoall(Tensor! dest, Tensor src, str group_name) -> ()");
  m.impl("nvshmem_alltoall", c10::kCUDA, &alltoall);
  m.impl("nvshmem_alltoall", c10::kMeta, &fake_alltoall);
  m.def("nvshmem_sum_reduce(Tensor! dest, Tensor src, int nelems, str group_name) -> ()");
  m.impl("nvshmem_sum_reduce", c10::kCUDA, &sum_reduce);
  m.impl("nvshmem_sum_reduce", c10::kMeta, &fake_sum_reduce);
  m.def(
      "nvshmem_allreduce_on_stream_with_copy(Tensor! dest_symm, Tensor source_symm, Tensor "
      "dest_local, Tensor source_local, int nelems, str group_name) -> ()");
  m.impl("nvshmem_allreduce_on_stream_with_copy", c10::kCUDA, &allreduce_on_stream_with_copy);
  m.impl("nvshmem_allreduce_on_stream_with_copy", c10::kMeta, &fake_allreduce_on_stream_with_copy);
};

}  // namespace
